#include "hip/hip_runtime.h"
#include "GPUSpikingModel.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Connections.h"
#include "Global.h"
#include "IAllNeurons.h"
#include "IAllSynapses.h"

#ifdef PERFORMANCE_METRICS
float g_time;
hipEvent_t start, stop;
#endif // PERFORMANCE_METRICS

__constant__ int d_debug_mask[1];

GPUSpikingModel::GPUSpikingModel() :
  Model::Model(),
  synapseIndexMapDevice_(NULL),
  randNoise_d(NULL),
  allNeuronsDevice_(NULL),
  allSynapsesDevice_(NULL)
{
}

GPUSpikingModel::~GPUSpikingModel() 
{
  //Let Model base class handle de-allocation
}

/// Allocates  and initializes memories on CUDA device.
/// @param[out] allNeuronsDevice          Memory location of the pointer to the neurons list on device memory.
/// @param[out] allSynapsesDevice         Memory location of the pointer to the synapses list on device memory.
void GPUSpikingModel::allocDeviceStruct(void** allNeuronsDevice, void** allSynapsesDevice)
{
  // Get neurons and synapses
  shared_ptr<IAllNeurons> neurons = layout_->getNeurons();
  shared_ptr<IAllSynapses> synapses = connections_->getSynapses();

  // Allocate Neurons and Synapses structs on GPU device memory
  neurons->allocNeuronDeviceStruct(allNeuronsDevice);
  synapses->allocSynapseDeviceStruct(allSynapsesDevice);

  // Allocate memory for random noise array
  int numNeurons = Simulator::getInstance().getTotalNeurons();
  BGSIZE randNoise_d_size = numNeurons * sizeof (float);	// size of random noise array
  HANDLE_ERROR( hipMalloc ( ( void ** ) &randNoise_d, randNoise_d_size ) );

  // Copy host neuron and synapse arrays into GPU device
  neurons->copyNeuronHostToDevice( *allNeuronsDevice );
  synapses->copySynapseHostToDevice( *allSynapsesDevice );

  // Allocate synapse inverse map in device memory
  allocSynapseImap( numNeurons );
}

/// Copies device memories to host memories and deallocates them.
/// @param[out] allNeuronsDevice          Memory location of the pointer to the neurons list on device memory.
/// @param[out] allSynapsesDevice         Memory location of the pointer to the synapses list on device memory.
void GPUSpikingModel::deleteDeviceStruct(void** allNeuronsDevice, void** allSynapsesDevice)
{  
  // Get neurons and synapses
  shared_ptr<IAllNeurons> neurons = layout_->getNeurons();
  shared_ptr<IAllSynapses> synapses = connections_->getSynapses();

  // Copy device synapse and neuron structs to host memory
  neurons->copyNeuronDeviceToHost( *allNeuronsDevice);
  // Deallocate device memory
  neurons->deleteNeuronDeviceStruct( *allNeuronsDevice);
  // Copy device synapse and neuron structs to host memory
  synapses->copySynapseDeviceToHost( *allSynapsesDevice);
  // Deallocate device memory
  synapses->deleteSynapseDeviceStruct( *allSynapsesDevice );
  HANDLE_ERROR( hipFree( randNoise_d ) );
}

/// Sets up the Simulation.
void GPUSpikingModel::setupSim()
{
  // Set device ID
  HANDLE_ERROR( hipSetDevice( g_deviceId ) );
  // Set DEBUG flag
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(d_debug_mask), &g_debug_mask, sizeof(int) ) );
  Model::setupSim();

  //initialize Mersenne Twister
  //assuming numNeurons >= 100 and is a multiple of 100. Note rng_mt_rng_count must be <= MT_RNG_COUNT
  int rng_blocks = 25; //# of blocks the kernel will use
  int rng_nPerRng = 4; //# of iterations per thread (thread granularity, # of rands generated per thread)
  int rng_mt_rng_count = Simulator::getInstance().getTotalNeurons() / rng_nPerRng; //# of threads to generate for numNeurons rand #s
  int rng_threads = rng_mt_rng_count/rng_blocks; //# threads per block needed
  initMTGPU(Simulator::getInstance().getSeed(), rng_blocks, rng_threads, rng_nPerRng, rng_mt_rng_count);

#ifdef PERFORMANCE_METRICS
  hipEventCreate( &start );
  hipEventCreate( &stop );

  t_gpu_rndGeneration = 0.0;
  t_gpu_advanceNeurons = 0.0;
  t_gpu_advanceSynapses = 0.0;
  t_gpu_calcSummation = 0.0;
#endif // PERFORMANCE_METRICS

  // allocates memories on CUDA device
  allocDeviceStruct((void **)&allNeuronsDevice_, (void **)&allSynapsesDevice_);

  // copy inverse map to the device memory
  copySynapseIndexMapHostToDevice(*(connections_->getSynapseIndexMap().get()), Simulator::getInstance().getTotalNeurons());

  // set some parameters used for advanceNeuronsDevice
  layout_->getNeurons()->setAdvanceNeuronsDeviceParams(*(connections_->getSynapses().get()));

  // set some parameters used for advanceSynapsesDevice
  connections_->getSynapses()->setAdvanceSynapsesDeviceParams();
}

/// Performs any finalization tasks on network following a simulation.
void GPUSpikingModel::finish()
{
  // deallocates memories on CUDA device
  deleteDeviceStruct((void**)&allNeuronsDevice_, (void**)&allSynapsesDevice_);
  deleteSynapseImap();

#ifdef PERFORMANCE_METRICS
  hipEventDestroy( start );
  hipEventDestroy( stop );
#endif // PERFORMANCE_METRICS
}

/// Advance everything in the model one time step. In this case, that
/// means calling all of the kernels that do the "micro step" updating
/// (i.e., NOT the stuff associated with growth).
void GPUSpikingModel::advance()
{
#ifdef PERFORMANCE_METRICS
  // Reset CUDA timer to start measurement of GPU operations
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // Get neurons and synapses
  shared_ptr<IAllNeurons> neurons = layout_->getNeurons();
  shared_ptr<IAllSynapses> synapses = connections_->getSynapses();

  normalMTGPU(randNoise_d);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_rndGeneration);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // display running info to console
  // Advance neurons ------------->
   dynamic_cast<AllSpikingNeurons *>(neurons.get())->advanceNeurons(*(connections_->getSynapses().get()), allNeuronsDevice_, allSynapsesDevice_, randNoise_d, synapseIndexMapDevice_);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_advanceNeurons);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // Advance synapses ------------->
  synapses->advanceSynapses(allSynapsesDevice_, allNeuronsDevice_, synapseIndexMapDevice_);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_advanceSynapses);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // calculate summation point
  calcSummationMap();

#ifdef PERFORMANCE_METRICS
 cudaLapTime(t_gpu_calcSummation);
#endif // PERFORMANCE_METRICS
}

/// Add psr of all incoming synapses to summation points.
void GPUSpikingModel::calcSummationMap()
{
  // CUDA parameters
  const int threadsPerBlock = 256;
  int blocksPerGrid = ( Simulator::getInstance().getTotalNeurons() + threadsPerBlock - 1 ) / threadsPerBlock;

  calcSummationMapDevice <<< blocksPerGrid, threadsPerBlock >>> (
        Simulator::getInstance().getTotalNeurons(), allNeuronsDevice_, synapseIndexMapDevice_, allSynapsesDevice_ );
}

/// Update the connection of all the Neurons and Synapses of the simulation.
void GPUSpikingModel::updateConnections()
{
  // Get neurons and synapses
  shared_ptr<IAllNeurons> neurons = layout_->getNeurons();
  shared_ptr<IAllSynapses> synapses = connections_->getSynapses();

  dynamic_cast<AllSpikingNeurons*>(neurons.get())->copyNeuronDeviceSpikeCountsToHost(allNeuronsDevice_);
  dynamic_cast<AllSpikingNeurons*>(neurons.get())->copyNeuronDeviceSpikeHistoryToHost(allNeuronsDevice_);

  // Update Connections data
  if (connections_->updateConnections(*(neurons.get()), layout_.get())) {
    connections_->updateSynapsesWeights(Simulator::getInstance().getTotalNeurons(), *(neurons.get()), *(synapses.get()), allNeuronsDevice_, allSynapsesDevice_, layout_.get());
    // create synapse index map
    connections_->createSynapseIndexMap();
    // copy index map to the device memory
    copySynapseIndexMapHostToDevice(*(connections_->getSynapseIndexMap().get()), Simulator::getInstance().getTotalNeurons());
  }
}

/// Update the Neuron's history.
void GPUSpikingModel::updateHistory()
{
  Model::updateHistory();
  // clear spike count
  
  shared_ptr<IAllNeurons> neurons = layout_->getNeurons();
  dynamic_cast<AllSpikingNeurons*>(neurons.get())->clearNeuronSpikeCounts(allNeuronsDevice_);
}

/// Allocate device memory for synapse inverse map.
/// @param  count	The number of neurons.
void GPUSpikingModel::allocSynapseImap( int count )
{
  SynapseIndexMap synapseIMapDevice;

  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.outgoingSynapseBegin_, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.outgoingSynapseCount_, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIMapDevice.outgoingSynapseBegin_, 0, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIMapDevice.outgoingSynapseCount_, 0, count * sizeof( BGSIZE ) ) );

  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.incomingSynapseBegin_, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.incomingSynapseCount_, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIMapDevice.incomingSynapseBegin_, 0, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIMapDevice.incomingSynapseCount_, 0, count * sizeof( BGSIZE ) ) );

  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMapDevice_, sizeof( SynapseIndexMap ) ) );
  HANDLE_ERROR( hipMemcpy( synapseIndexMapDevice_, &synapseIMapDevice, sizeof( SynapseIndexMap ), 
        hipMemcpyHostToDevice ) );
}

/// Deallocate device memory for synapse inverse map.
void GPUSpikingModel::deleteSynapseImap(  )
{
  SynapseIndexMap synapseIMapDevice;

  HANDLE_ERROR( hipMemcpy ( &synapseIMapDevice, synapseIndexMapDevice_, 
        sizeof( SynapseIndexMap ), hipMemcpyDeviceToHost ) );

  HANDLE_ERROR( hipFree( synapseIMapDevice.outgoingSynapseBegin_ ) );
  HANDLE_ERROR( hipFree( synapseIMapDevice.outgoingSynapseCount_ ) );
  HANDLE_ERROR( hipFree( synapseIMapDevice.outgoingSynapseIndexMap_ ) );

  HANDLE_ERROR( hipFree( synapseIMapDevice.incomingSynapseBegin_ ) );
  HANDLE_ERROR( hipFree( synapseIMapDevice.incomingSynapseCount_ ) );
  HANDLE_ERROR( hipFree( synapseIMapDevice.incomingSynapseIndexMap_ ) );

  HANDLE_ERROR( hipFree( synapseIndexMapDevice_ ) );
}

/// Copy SynapseIndexMap in host memory to SynapseIndexMap in device memory.
/// @param  synapseIndexMapHost		Reference to the SynapseIndexMap in host memory.
void GPUSpikingModel::copySynapseIndexMapHostToDevice(SynapseIndexMap &synapseIndexMapHost, int numNeurons)
{
  shared_ptr<IAllSynapses> synapses = connections_->getSynapses();
  int totalSynapseCount = dynamic_cast<AllSynapses*>(synapses.get())->totalSynapseCount_;

  if (totalSynapseCount == 0)
    return;

  // TODO: rename variable, DevicePointer
  SynapseIndexMap synapseIMapDevice;

  HANDLE_ERROR( hipMemcpy ( &synapseIMapDevice, synapseIndexMapDevice_, 
        sizeof( SynapseIndexMap ), hipMemcpyDeviceToHost ) );

  // forward map
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.outgoingSynapseBegin_, 
        synapseIndexMapHost.outgoingSynapseBegin_, numNeurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.outgoingSynapseCount_, 
        synapseIndexMapHost.outgoingSynapseCount_, numNeurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  // the number of synapses may change, so we reallocate the memory
  if (synapseIMapDevice.outgoingSynapseIndexMap_ != NULL) {
    HANDLE_ERROR( hipFree( synapseIMapDevice.outgoingSynapseIndexMap_ ) );
  }
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.outgoingSynapseIndexMap_, 
        totalSynapseCount * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.outgoingSynapseIndexMap_, synapseIndexMapHost.outgoingSynapseIndexMap_, 
        totalSynapseCount * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );

  // active synapse map
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.incomingSynapseBegin_, 
        synapseIndexMapHost.incomingSynapseBegin_, numNeurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.incomingSynapseCount_, 
        synapseIndexMapHost.incomingSynapseCount_, numNeurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  // the number of synapses may change, so we reallocate the memory
  if (synapseIMapDevice.incomingSynapseIndexMap_ != NULL) {
    HANDLE_ERROR( hipFree( synapseIMapDevice.incomingSynapseIndexMap_ ) );
  }
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIMapDevice.incomingSynapseIndexMap_, 
        totalSynapseCount * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemcpy ( synapseIMapDevice.incomingSynapseIndexMap_, synapseIndexMapHost.incomingSynapseIndexMap_, 
        totalSynapseCount * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );

  HANDLE_ERROR( hipMemcpy ( synapseIndexMapDevice_, &synapseIMapDevice, 
        sizeof( SynapseIndexMap ), hipMemcpyHostToDevice ) );
}

/**
 * Calculate the sum of synaptic input to each neuron.
 *
 * Calculate the sum of synaptic input to each neuron. One thread
 * corresponds to one neuron. Iterates sequentially through the
 * forward synapse index map (synapseIndexMapDevice_) to access only
 * existing synapses. Using this structure eliminates the need to skip
 * synapses that have undergone lazy deletion from the main
 * (allSynapsesDevice) synapse structure. The forward map is
 * re-computed during each network restructure (once per epoch) to
 * ensure that all synapse pointers for a neuron are stored
 * contiguously.
 * 
 * @param[in] totalNeurons           Number of neurons in the entire simulation.
 * @param[in,out] allNeuronsDevice   Pointer to Neuron structures in device memory.
 * @param[in] synapseIndexMapDevice_  Pointer to forward map structures in device memory.
 * @param[in] allSynapsesDevice      Pointer to Synapse structures in device memory.
 */
__global__ void calcSummationMapDevice(int totalNeurons, 
				       AllSpikingNeuronsDeviceProperties* __restrict__ allNeuronsDevice, 
				       const SynapseIndexMap* __restrict__ synapseIndexMapDevice_, 
				       const AllSpikingSynapsesDeviceProperties* __restrict__ allSynapsesDevice)
{
  // The usual thread ID calculation and guard against excess threads
  // (beyond the number of neurons, in this case).
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if ( idx >= totalNeurons )
    return;

  // Number of incoming synapses
  const BGSIZE synCount = synapseIndexMapDevice_->incomingSynapseCount_[idx];
  // Optimization: terminate thread if no incoming synapses
  if (synCount != 0) {
    // Index of start of this neuron's block of forward map entries
    const int beginIndex = synapseIndexMapDevice_->incomingSynapseBegin_[idx];
    // Address of the start of this neuron's block of forward map entries
    const BGSIZE* activeMapBegin = 
      &(synapseIndexMapDevice_->incomingSynapseIndexMap_[beginIndex]);
    // Summed post-synaptic response (PSR)
    BGFLOAT sum = 0.0;
    // Index of the current incoming synapse
    BGSIZE synIndex;
    // Repeat for each incoming synapse
    for (BGSIZE i = 0; i < synCount; i++) {
      // Get index of current incoming synapse
      synIndex = activeMapBegin[i];
      // Fetch its PSR and add into sum
      sum += allSynapsesDevice->psr_[synIndex];
    }
    // Store summed PSR into this neuron's summation point
    allNeuronsDevice->summationMap_[idx] = sum;
  }
}

/// Copy GPU Synapse data to CPU.
void GPUSpikingModel::copyGPUtoCPU()
{
  // copy device synapse structs to host memory
  connections_->getSynapses()->copySynapseDeviceToHost(allSynapsesDevice_);
}

/// Copy CPU Synapse data to GPU.
void GPUSpikingModel::copyCPUtoGPU()
{
  // copy host synapse structs to device memory
  connections_->getSynapses()->copySynapseHostToDevice(allSynapsesDevice_);
}

/// Print out SynapseProps on the GPU.
void GPUSpikingModel::printGPUSynapsesPropsModel() const
{  
  connections_->getSynapses()->printGPUSynapsesProps(allSynapsesDevice_);
}

