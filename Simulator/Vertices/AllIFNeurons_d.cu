/*
 * AllIFNeurons_d.cu
 *
 */

#include "AllIFNeurons.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice ) {
	AllIFNeuronsDeviceProperties allNeurons;

	allocDeviceStruct( allNeurons );

        HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeuronsDevice         GPU address of the AllIFNeuronsDeviceProperties struct.
 */
void AllIFNeurons::allocDeviceStruct( AllIFNeuronsDeviceProperties &allNeuronsDevice ) {
	int count = Simulator::getInstance().getTotalNeurons();
	int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.C1_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.C2_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Cm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.I0_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Iinject_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Inoise_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Isyn_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Rm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Tau_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Trefract_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Vinit_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Vm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Vreset_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Vrest_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.Vthresh_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.hasFired_, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.numStepsInRefractoryPeriod_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.spikeCount_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.spikeCountOffset_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.summationMap_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice.spikeHistory_, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count];
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &pSpikeHistory[i], maxSpikes * sizeof( uint64_t ) ) );
	}
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.spikeHistory_, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	// get device summation point address
	summationMap_ = allNeuronsDevice.summationMap_;
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice ) {
	AllIFNeuronsDeviceProperties allNeuronsDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allNeuronsDeviceProps, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeuronsDeviceProps );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeuronsDevice         GPU address of the AllIFNeuronsDeviceProperties struct.
 */
void AllIFNeurons::deleteDeviceStruct( AllIFNeuronsDeviceProperties& allNeuronsDevice ) {
	int count = Simulator::getInstance().getTotalNeurons();

	uint64_t* pSpikeHistory[count];
	HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeuronsDevice.spikeHistory_,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipFree( pSpikeHistory[i] ) );
	}

	HANDLE_ERROR( hipFree( allNeuronsDevice.C1_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.C2_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Cm_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.I0_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Iinject_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Inoise_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Isyn_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Rm_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Tau_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Trefract_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Vinit_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Vm_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Vreset_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Vrest_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.Vthresh_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.hasFired_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.numStepsInRefractoryPeriod_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.spikeCount_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.spikeCountOffset_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.summationMap_ ) );
	HANDLE_ERROR( hipFree( allNeuronsDevice.spikeHistory_ ) );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::copyNeuronHostToDevice( void* allNeuronsDevice ) { 
	AllIFNeuronsDeviceProperties allNeuronsDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allNeuronsDeviceProps, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeuronsDeviceProps );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeuronsDevice         GPU address of the AllIFNeuronsDeviceProperties struct.
 */
void AllIFNeurons::copyHostToDevice( AllIFNeuronsDeviceProperties& allNeuronsDevice ) { 
	int count = Simulator::getInstance().getTotalNeurons();

	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.C1_, C1_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.C2_, C2_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Cm_, Cm_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.I0_, I0_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Iinject_, Iinject_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Inoise_, Inoise_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Isyn_, Isyn_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Rm_, Rm_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Tau_, Tau_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Trefract_, Trefract_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Vinit_, Vinit_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Vm_, Vm_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Vreset_, Vreset_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Vrest_, Vrest_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.Vthresh_, Vthresh_, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.hasFired_, hasFired_, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.numStepsInRefractoryPeriod_, numStepsInRefractoryPeriod_, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.spikeCount_, spikeCount_, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsDevice.spikeCountOffset_, spikeCountOffset_, count * sizeof( int ), hipMemcpyHostToDevice ) );

        int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
        uint64_t* pSpikeHistory[count];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeuronsDevice.spikeHistory_, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                HANDLE_ERROR( hipMemcpy ( pSpikeHistory[i], spikeHistory_[i], maxSpikes * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        }
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice ) {
	AllIFNeuronsDeviceProperties allNeuronsDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allNeuronsDeviceProps, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeuronsDeviceProps );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeuronsDevice         GPU address of the AllIFNeuronsDeviceProperties struct.
 */
void AllIFNeurons::copyDeviceToHost( AllIFNeuronsDeviceProperties& allNeuronsDevice ) {
	int count = Simulator::getInstance().getTotalNeurons();

	HANDLE_ERROR( hipMemcpy ( C1_, allNeuronsDevice.C1_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C2_, allNeuronsDevice.C2_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cm_, allNeuronsDevice.Cm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( I0_, allNeuronsDevice.I0_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Iinject_, allNeuronsDevice.Iinject_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Inoise_, allNeuronsDevice.Inoise_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Isyn_, allNeuronsDevice.Isyn_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Rm_, allNeuronsDevice.Rm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Tau_, allNeuronsDevice.Tau_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Trefract_, allNeuronsDevice.Trefract_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vinit_, allNeuronsDevice.Vinit_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vm_, allNeuronsDevice.Vm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vreset_, allNeuronsDevice.Vreset_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vrest_, allNeuronsDevice.Vrest_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vthresh_, allNeuronsDevice.Vthresh_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( hasFired_, allNeuronsDevice.hasFired_, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( numStepsInRefractoryPeriod_, allNeuronsDevice.numStepsInRefractoryPeriod_, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCount_, allNeuronsDevice.spikeCount_, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCountOffset_, allNeuronsDevice.spikeCountOffset_, count * sizeof( int ), hipMemcpyDeviceToHost ) );

        int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
        uint64_t* pSpikeHistory[count];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeuronsDevice.spikeHistory_, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                HANDLE_ERROR( hipMemcpy ( spikeHistory_[i], pSpikeHistory[i], maxSpikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice ) 
{        
        AllIFNeuronsDeviceProperties allNeuronsDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allNeuronsDeviceProps, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );        
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeuronsDeviceProps );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice )
{
        AllIFNeuronsDeviceProperties allNeuronsDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allNeuronsDeviceProps, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeuronsDeviceProps );
}

/*
 *  Clear the spike counts out of all neurons.
 *
 *  @param  allNeuronsDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
 */
void AllIFNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice )
{
        AllIFNeuronsDeviceProperties allNeuronsDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allNeuronsDeviceProps, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeuronsDeviceProps );
}


/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsDevice       GPU address of the AllIFNeuronsDeviceProperties struct 
 *                                 on device memory.
 *  @param  allSynapsesDevice      GPU address of the allSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  GPU address of the SynapseIndexMap on device memory.
 */
void AllIFNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsDevice, void* allSynapsesDevice, float* randNoise, SynapseIndexMap* synapseIndexMapDevice )
{
}
