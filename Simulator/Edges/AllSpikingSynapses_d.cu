/*
 * AllSpikingSynapses.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice ) {
        allocSynapseDeviceStruct( allSynapsesDevice, Simulator::getInstance().getTotalNeurons(), Simulator::getInstance().getMaxSynapsesPerNeuron() );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        allocDeviceStruct( allSynapses, numNeurons, maxSynapsesPerNeuron );

        HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocDeviceStruct( AllSpikingSynapsesDeviceProperties &allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) {
        BGSIZE maxTotalSynapses = maxSynapsesPerNeuron * numNeurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.sourceNeuronIndex_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.destNeuronIndex_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.W_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.type_, maxTotalSynapses * sizeof( synapseType ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.psr_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.inUse_, maxTotalSynapses * sizeof( bool ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.synapseCounts_, numNeurons * sizeof( BGSIZE ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.decay_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.tau_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.totalDelay_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.delayQueue_, maxTotalSynapses * sizeof( uint32_t ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.delayIndex_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.delayQueueLength_, maxTotalSynapses * sizeof( int ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        deleteDeviceStruct( allSynapses );

        HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::deleteDeviceStruct( AllSpikingSynapsesDeviceProperties& allSynapsesDevice ) {
        HANDLE_ERROR( hipFree( allSynapsesDevice.sourceNeuronIndex_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.destNeuronIndex_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.W_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.type_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.psr_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.inUse_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.synapseCounts_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.decay_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.tau_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.totalDelay_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.delayQueue_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.delayIndex_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.delayQueueLength_ ) );

        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.countNeurons_ = 0;
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice ) { // copy everything necessary
        copySynapseHostToDevice( allSynapsesDevice, Simulator::getInstance().getTotalNeurons(), Simulator::getInstance().getMaxSynapsesPerNeuron() );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) { // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapsesDeviceProps;

        HANDLE_ERROR( hipMemcpy ( &allSynapsesDeviceProps, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyHostToDevice( allSynapsesDevice, allSynapsesDeviceProps, numNeurons, maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice           GPU address of the allSynapses struct on device memory.     
 *  @param  allSynapsesDeviceProps      GPU address of the AllSpikingSynapsesDeviceProperties struct on device memory.
 *  @param  numNeurons                  Number of neurons.
 *  @param  maxSynapsesPerNeuron        Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyHostToDevice( void* allSynapsesDevice, AllSpikingSynapsesDeviceProperties& allSynapsesDeviceProps, int numNeurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        BGSIZE maxTotalSynapses = maxSynapsesPerNeuron * numNeurons;

        allSynapsesDeviceProps.maxSynapsesPerNeuron_ = maxSynapsesPerNeuron_;
        allSynapsesDeviceProps.totalSynapseCount_ = totalSynapseCount_;
        allSynapsesDeviceProps.countNeurons_ = countNeurons_;
        HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapsesDeviceProps, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );

        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapsesDeviceProps.countNeurons_ = 0;

        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.sourceNeuronIndex_, sourceNeuronIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.destNeuronIndex_, destNeuronIndex_,
                maxTotalSynapses * sizeof( int ),  hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.W_, W_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.type_, type_,
                maxTotalSynapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.psr_, psr_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.inUse_, inUse_,
                maxTotalSynapses * sizeof( bool ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.synapseCounts_, synapseCounts_,
                        numNeurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.decay_, decay_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.tau_, tau_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.totalDelay_, totalDelay_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.delayQueue_, delayQueue_,
                maxTotalSynapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.delayIndex_, delayIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.delayQueueLength_, delayQueueLength_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::copySynapseDeviceToHost( void* allSynapsesDevice ) {
        // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyDeviceToHost( allSynapses );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyDeviceToHost( AllSpikingSynapsesDeviceProperties& allSynapsesDevice ) {
        int numNeurons = Simulator::getInstance().getTotalNeurons();
        BGSIZE maxTotalSynapses = Simulator::getInstance().getMaxSynapsesPerNeuron() * numNeurons;

        HANDLE_ERROR( hipMemcpy ( synapseCounts_, allSynapsesDevice.synapseCounts_,
                numNeurons * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuron_ = allSynapsesDevice.maxSynapsesPerNeuron_;
        totalSynapseCount_ = allSynapsesDevice.totalSynapseCount_;
        countNeurons_ = allSynapsesDevice.countNeurons_;

        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapsesDevice.countNeurons_ = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndex_, allSynapsesDevice.sourceNeuronIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndex_, allSynapsesDevice.destNeuronIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( W_, allSynapsesDevice.W_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( type_, allSynapsesDevice.type_,
                maxTotalSynapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psr_, allSynapsesDevice.psr_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUse_, allSynapsesDevice.inUse_,
                maxTotalSynapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( decay_, allSynapsesDevice.decay_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tau_, allSynapsesDevice.tau_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( totalDelay_, allSynapsesDevice.totalDelay_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueue_, allSynapsesDevice.delayQueue_,
                maxTotalSynapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayIndex_, allSynapsesDevice.delayIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueueLength_, allSynapsesDevice.delayQueueLength_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
}

/*
 *  Get synapse_counts in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::copyDeviceSynapseCountsToHost( void* allSynapsesDevice )
{
        AllSpikingSynapsesDeviceProperties allSynapsesDeviceProps;
        int neuronCount = Simulator::getInstance().getTotalNeurons();

        HANDLE_ERROR( hipMemcpy ( &allSynapsesDeviceProps, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapseCounts_, allSynapsesDeviceProps.synapseCounts_, neuronCount * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );

        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.countNeurons_ = 0;
}

/* 
 *  Get summationCoord and in_use in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSpikingSynapsesDeviceProperties struct
 *                             on device memory.
 */
void AllSpikingSynapses::copyDeviceSynapseSumIdxToHost(void* allSynapsesDevice )
{
        AllSpikingSynapsesDeviceProperties allSynapsesDeviceProps;
        BGSIZE maxTotalSynapses = Simulator::getInstance().getMaxSynapsesPerNeuron() * Simulator::getInstance().getTotalNeurons();

        HANDLE_ERROR( hipMemcpy ( &allSynapsesDeviceProps, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndex_, allSynapsesDeviceProps.sourceNeuronIndex_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUse_, allSynapsesDeviceProps.inUse_,
                maxTotalSynapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
       
        // Set countNeurons_ to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.countNeurons_ = 0;
}

/*
 *  Set some parameters used for advanceSynapsesDevice.
 */
void AllSpikingSynapses::setAdvanceSynapsesDeviceParams()
{
    setSynapseClassID();
}

/**
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllSpikingSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllSpikingSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses) ) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      GPU address of the AllSynapsesDeviceProperties struct
 *                                 on device memory.
 *  @param  allNeuronsDevice       GPU address of the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  GPU address of the SynapseIndexMap on device memory.
 */
void AllSpikingSynapses::advanceSynapses(void* allSynapsesDevice, void* allNeuronsDevice, void* synapseIndexMapDevice )
{
    if (totalSynapseCount_ == 0)
        return;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( totalSynapseCount_ + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance synapses ------------->
    advanceSpikingSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( totalSynapseCount_, (SynapseIndexMap*) synapseIndexMapDevice, g_simulationStep, Simulator::getInstance().getDeltaT(), (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice );
}

/*
 *  Prints GPU SynapsesProps data.
 *   
 *  @param  allSynapsesDeviceProps   GPU address of the corresponding SynapsesDeviceProperties struct on device memory.
 */
void AllSpikingSynapses::printGPUSynapsesProps( void* allSynapsesDeviceProps ) const
{
    AllSpikingSynapsesDeviceProperties allSynapsesProps;

    //allocate print out data members
    BGSIZE size = maxSynapsesPerNeuron_ * countNeurons_;
    if (size != 0) {
        BGSIZE *synapseCountsPrint = new BGSIZE[countNeurons_];
        BGSIZE maxSynapsesPerNeuronPrint;
        BGSIZE totalSynapseCountPrint;
        int countNeuronsPrint;
        int *sourceNeuronIndexPrint = new int[size];
        int *destNeuronIndexPrint = new int[size];
        BGFLOAT *WPrint = new BGFLOAT[size];

        synapseType *typePrint = new synapseType[size];
        BGFLOAT *psrPrint = new BGFLOAT[size];
        bool *inUsePrint = new bool[size];

        for (BGSIZE i = 0; i < size; i++) {
            inUsePrint[i] = false;
        }

        for (int i = 0; i < countNeurons_; i++) {
            synapseCountsPrint[i] = 0;
        }

        BGFLOAT *decayPrint = new BGFLOAT[size];
        int *totalDelayPrint = new int[size];
        BGFLOAT *tauPrint = new BGFLOAT[size];


        // copy everything
        HANDLE_ERROR( hipMemcpy ( &allSynapsesProps, allSynapsesDeviceProps, sizeof( AllSpikingSynapsesDeviceProperties), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapseCountsPrint, allSynapsesProps.synapseCounts_, countNeurons_ * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuronPrint = allSynapsesProps.maxSynapsesPerNeuron_;
        totalSynapseCountPrint = allSynapsesProps.totalSynapseCount_;
        countNeuronsPrint = allSynapsesProps.countNeurons_;

        // Set countNeurons_ to 0 to avoid illegal memory deallocation
        // at AllSynapsesProps deconstructor.
        allSynapsesProps.countNeurons_ = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndexPrint, allSynapsesProps.sourceNeuronIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndexPrint, allSynapsesProps.destNeuronIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WPrint, allSynapsesProps.W_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( typePrint, allSynapsesProps.type_, size * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psrPrint, allSynapsesProps.psr_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUsePrint, allSynapsesProps.inUse_, size * sizeof( bool ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( decayPrint, allSynapsesProps.decay_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauPrint, allSynapsesProps.tau_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( totalDelayPrint, allSynapsesProps.totalDelay_, size * sizeof( int ), hipMemcpyDeviceToHost ) );


        for(int i = 0; i < maxSynapsesPerNeuron_ * countNeurons_; i++) {
            if (WPrint[i] != 0.0) {
                cout << "GPU W[" << i << "] = " << WPrint[i];
                cout << " GPU sourNeuron: " << sourceNeuronIndexPrint[i];
                cout << " GPU desNeuron: " << destNeuronIndexPrint[i];
                cout << " GPU type: " << typePrint[i];
                cout << " GPU psr: " << psrPrint[i];
                cout << " GPU in_use:" << inUsePrint[i];

                cout << " GPU decay: " << decayPrint[i];
                cout << " GPU tau: " << tauPrint[i];
                cout << " GPU total_delay: " << totalDelayPrint[i] << endl;;
            }
        }

        for (int i = 0; i < countNeurons_; i++) {
            cout << "GPU synapse_counts:" << "neuron[" << i  << "]" << synapseCountsPrint[i] << endl;
        }

        cout << "GPU totalSynapseCount:" << totalSynapseCountPrint << endl;
        cout << "GPU maxSynapsesPerNeuron:" << maxSynapsesPerNeuronPrint << endl;
        cout << "GPU countNeurons_:" << countNeuronsPrint << endl;


        // Set countNeurons_ to 0 to avoid illegal memory deallocation
        // at AllDSSynapsesProps deconstructor.
        allSynapsesProps.countNeurons_ = 0;

        delete[] destNeuronIndexPrint;
        delete[] WPrint;
        delete[] sourceNeuronIndexPrint;
        delete[] psrPrint;
        delete[] typePrint;
        delete[] inUsePrint;
        delete[] synapseCountsPrint;
        destNeuronIndexPrint = NULL;
        WPrint = NULL;
        sourceNeuronIndexPrint = NULL;
        psrPrint = NULL;
        typePrint = NULL;
        inUsePrint = NULL;
        synapseCountsPrint = NULL;

        delete[] decayPrint;
        delete[] totalDelayPrint;
        delete[] tauPrint;
        decayPrint = NULL;
        totalDelayPrint = NULL;
        tauPrint = NULL;
    }
}


