/*
 * AllDynamicSTDPSynapses_d.cu
 *
 */

#include "AllDynamicSTDPSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"
#include "Simulator.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  GPU address of the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllDynamicSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice ) {
	allocSynapseDeviceStruct( allSynapsesDevice, Simulator::getInstance().getTotalNeurons(), Simulator::getInstance().getMaxSynapsesPerNeuron() );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     GPU address of the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) {
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

	allocDeviceStruct( allSynapses, numNeurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDeviceProps      GPU address of the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                      on device memory.
 *  @param  numNeurons                  Number of neurons.
 *  @param  maxSynapsesPerNeuron        Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::allocDeviceStruct( AllDynamicSTDPSynapsesDeviceProperties &allSynapsesDeviceProps, int numNeurons, int maxSynapsesPerNeuron ) {
        AllSTDPSynapses::allocDeviceStruct( allSynapsesDeviceProps, numNeurons, maxSynapsesPerNeuron );

        BGSIZE maxTotalSynapses = maxSynapsesPerNeuron * numNeurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDeviceProps.lastSpike_, maxTotalSynapses * sizeof( uint64_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDeviceProps.r_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDeviceProps.u_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDeviceProps.D_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDeviceProps.U_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDeviceProps.F_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  GPU address of the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllDynamicSTDPSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDeviceProps  GPU address of the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                  on device memory.
 */
void AllDynamicSTDPSynapses::deleteDeviceStruct( AllDynamicSTDPSynapsesDeviceProperties& allSynapsesDeviceProps ) {
        HANDLE_ERROR( hipFree( allSynapsesDeviceProps.lastSpike_ ) );
	HANDLE_ERROR( hipFree( allSynapsesDeviceProps.r_ ) );
	HANDLE_ERROR( hipFree( allSynapsesDeviceProps.u_ ) );
	HANDLE_ERROR( hipFree( allSynapsesDeviceProps.D_ ) );
	HANDLE_ERROR( hipFree( allSynapsesDeviceProps.U_ ) );
	HANDLE_ERROR( hipFree( allSynapsesDeviceProps.F_ ) );

        AllSTDPSynapses::deleteDeviceStruct( allSynapsesDeviceProps );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  GPU address of the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllDynamicSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, Simulator::getInstance().getTotalNeurons(), Simulator::getInstance().getMaxSynapsesPerNeuron() );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     GPU address of the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allSynapsesDevice, allSynapses, numNeurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice           GPU address of the allSynapses struct on device memory.
 *  @param  allSynapsesDeviceProps      GPU address of the allDynamicSTDPSSynapses struct on device memory.
 *  @param  numNeurons                  Number of neurons.
 *  @param  maxSynapsesPerNeuron        Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copyHostToDevice( void* allSynapsesDevice, AllDynamicSTDPSynapsesDeviceProperties& allSynapsesDeviceProps, int numNeurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSTDPSynapses::copyHostToDevice( allSynapsesDevice, allSynapsesDeviceProps, numNeurons, maxSynapsesPerNeuron );

        BGSIZE maxTotalSynapses = maxSynapsesPerNeuron * numNeurons;
        
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.lastSpike_, lastSpike_,
                maxTotalSynapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.r_, r_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.u_, u_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.D_, D_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.U_, U_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.F_, F_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  GPU address of the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllDynamicSTDPSynapses::copySynapseDeviceToHost( void* allSynapsesDevice ) {
	// copy everything necessary
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice           GPU address of the allSynapses struct on device memory.
 *  @param  allSynapsesDeviceProps      GPU address of the allDynamicSTDPSSynapses struct on device memory.
 *  @param  numNeurons                  Number of neurons.
 *  @param  maxSynapsesPerNeuron        Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copyDeviceToHost( AllDynamicSTDPSynapsesDeviceProperties& allSynapsesDeviceProps ) {
        AllSTDPSynapses::copyDeviceToHost( allSynapsesDeviceProps ) ;

	int numNeurons = Simulator::getInstance().getTotalNeurons();
	BGSIZE maxTotalSynapses = Simulator::getInstance().getMaxSynapsesPerNeuron() * numNeurons;

        HANDLE_ERROR( hipMemcpy ( lastSpike_, allSynapsesDeviceProps.lastSpike_,
                maxTotalSynapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( r_, allSynapsesDeviceProps.r_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( u_, allSynapsesDeviceProps.u_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( D_, allSynapsesDeviceProps.D_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( U_, allSynapsesDeviceProps.U_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( F_, allSynapsesDeviceProps.F_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/**     
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllDynamicSTDPSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllDynamicSTDPSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

/*
 *  Prints GPU SynapsesProps data.
 *   
 *  @param  allSynapsesDeviceProps   GPU address of the corresponding SynapsesDeviceProperties struct on device memory.
 */
void AllDynamicSTDPSynapses::printGPUSynapsesProps( void* allSynapsesDeviceProps ) const
{
    AllDynamicSTDPSynapsesDeviceProperties allSynapsesProps;

    //allocate print out data members
    BGSIZE size = maxSynapsesPerNeuron_ * countNeurons_;
    if (size != 0) {
        BGSIZE *synapseCountsPrint = new BGSIZE[countNeurons_];
        BGSIZE maxSynapsesPerNeuronPrint;
        BGSIZE totalSynapseCountPrint;
        int countNeuronsPrint;
        int *sourceNeuronIndexPrint = new int[size];
        int *destNeuronIndexPrint = new int[size];
        BGFLOAT *WPrint = new BGFLOAT[size];

        synapseType *typePrint = new synapseType[size];
        BGFLOAT *psrPrint = new BGFLOAT[size];
        bool *inUsePrint = new bool[size];

        for (BGSIZE i = 0; i < size; i++) {
            inUsePrint[i] = false;
        }

        for (int i = 0; i < countNeurons_; i++) {
            synapseCountsPrint[i] = 0;
        }

        BGFLOAT *decayPrint = new BGFLOAT[size];
        int *totalDelayPrint = new int[size];
        BGFLOAT *tauPrint = new BGFLOAT[size];

        int *totalDelayPostPrint = new int[size];
        BGFLOAT *tauspost_Print = new BGFLOAT[size];
        BGFLOAT *tauspre_Print = new BGFLOAT[size];
        BGFLOAT *taupos_Print = new BGFLOAT[size];
        BGFLOAT *tauneg_Print = new BGFLOAT[size];
        BGFLOAT *STDPgap_Print = new BGFLOAT[size];
        BGFLOAT *Wex_Print = new BGFLOAT[size];
        BGFLOAT *Aneg_Print = new BGFLOAT[size];
        BGFLOAT *Apos_Print = new BGFLOAT[size];
        BGFLOAT *mupos_Print = new BGFLOAT[size];
        BGFLOAT *muneg_Print = new BGFLOAT[size];
        bool *useFroemkeDanSTDP_Print = new bool[size];

        uint64_t *lastSpikePrint = new uint64_t[size];
        BGFLOAT *rPrint = new BGFLOAT[size];
        BGFLOAT *uPrint = new BGFLOAT[size];
        BGFLOAT *DPrint = new BGFLOAT[size];
        BGFLOAT *UPrint = new BGFLOAT[size];
        BGFLOAT *FPrint = new BGFLOAT[size];

        // copy everything
        HANDLE_ERROR( hipMemcpy ( &allSynapsesProps, allSynapsesDeviceProps, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapseCountsPrint, allSynapsesProps.synapseCounts_, countNeurons_ * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuronPrint = allSynapsesProps.maxSynapsesPerNeuron_;
        totalSynapseCountPrint = allSynapsesProps.totalSynapseCount_;
        countNeuronsPrint = allSynapsesProps.countNeurons_;

        // Set countNeurons_ to 0 to avoid illegal memory deallocation
        // at AllSynapsesProps deconstructor.
        allSynapsesProps.countNeurons_ = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndexPrint, allSynapsesProps.sourceNeuronIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndexPrint, allSynapsesProps.destNeuronIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WPrint, allSynapsesProps.W_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( typePrint, allSynapsesProps.type_, size * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psrPrint, allSynapsesProps.psr_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUsePrint, allSynapsesProps.inUse_, size * sizeof( bool ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( decayPrint, allSynapsesProps.decay_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauPrint, allSynapsesProps.tau_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( totalDelayPrint, allSynapsesProps.totalDelay_,size * sizeof( int ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( totalDelayPostPrint, allSynapsesProps.totalDelayPost_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspost_Print, allSynapsesProps.tauspost_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspre_Print, allSynapsesProps.tauspre_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( taupos_Print, allSynapsesProps.taupos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauneg_Print, allSynapsesProps.tauneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( STDPgap_Print, allSynapsesProps.STDPgap_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Wex_Print, allSynapsesProps.Wex_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Aneg_Print, allSynapsesProps.Aneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Apos_Print, allSynapsesProps.Apos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( mupos_Print, allSynapsesProps.mupos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( muneg_Print, allSynapsesProps.muneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( useFroemkeDanSTDP_Print, allSynapsesProps.useFroemkeDanSTDP_, size * sizeof( bool ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( lastSpikePrint, allSynapsesProps.lastSpike_, size * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( rPrint, allSynapsesProps.r_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( uPrint, allSynapsesProps.u_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( DPrint, allSynapsesProps.D_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( UPrint, allSynapsesProps.U_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( FPrint, allSynapsesProps.F_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );

        for(int i = 0; i < maxSynapsesPerNeuron_ * countNeurons_; i++) {
            if (WPrint[i] != 0.0) {
                cout << "GPU W[" << i << "] = " << WPrint[i];
                cout << " GPU sourNeuron: " << sourceNeuronIndexPrint[i];
                cout << " GPU desNeuron: " << destNeuronIndexPrint[i];
                cout << " GPU type: " << typePrint[i];
                cout << " GPU psr: " << psrPrint[i];
                cout << " GPU in_use:" << inUsePrint[i];

                cout << " GPU decay: " << decayPrint[i];
                cout << " GPU tau: " << tauPrint[i];
                cout << " GPU total_delay: " << totalDelayPrint[i];

                cout << " GPU total_delayPost: " << totalDelayPostPrint[i];
                cout << " GPU tauspost_: " << tauspost_Print[i];
                cout << " GPU tauspre_: " << tauspre_Print[i];
                cout << " GPU taupos_: " << taupos_Print[i];
                cout << " GPU tauneg_: " << tauneg_Print[i];
                cout << " GPU STDPgap_: " << STDPgap_Print[i];
                cout << " GPU Wex_: " << Wex_Print[i];
                cout << " GPU Aneg_: " << Aneg_Print[i];
                cout << " GPU Apos_: " << Apos_Print[i];
                cout << " GPU mupos_: " << mupos_Print[i];
                cout << " GPU muneg_: " << muneg_Print[i];
                cout << " GPU useFroemkeDanSTDP_: " << useFroemkeDanSTDP_Print[i];

                cout << " GPU lastSpike: " << lastSpikePrint[i];
                cout << " GPU r: " << rPrint[i];
                cout << " GPU u: " << uPrint[i];
                cout << " GPU D: " << DPrint[i];
                cout << " GPU U: " << UPrint[i];
                cout << " GPU F: " << FPrint[i] << endl;
            }
        }

        for (int i = 0; i < countNeurons_; i++) {
            cout << "GPU synapse_counts:" << "neuron[" << i  << "]" << synapseCountsPrint[i] << endl;
        }

        cout << "GPU totalSynapseCount:" << totalSynapseCountPrint << endl;
        cout << "GPU maxSynapsesPerNeuron:" << maxSynapsesPerNeuronPrint << endl;
        cout << "GPU countNeurons_:" << countNeuronsPrint << endl;


        // Set countNeurons_ to 0 to avoid illegal memory deallocation
        // at AllDSSynapsesProps deconstructor.
        allSynapsesProps.countNeurons_ = 0;

        delete[] destNeuronIndexPrint;
        delete[] WPrint;
        delete[] sourceNeuronIndexPrint;
        delete[] psrPrint;
        delete[] typePrint;
        delete[] inUsePrint;
        delete[] synapseCountsPrint;
        destNeuronIndexPrint = NULL;
        WPrint = NULL;
        sourceNeuronIndexPrint = NULL;
        psrPrint = NULL;
        typePrint = NULL;
        inUsePrint = NULL;
        synapseCountsPrint = NULL;

        delete[] decayPrint;
        delete[] totalDelayPrint;
        delete[] tauPrint;
        decayPrint = NULL;
        totalDelayPrint = NULL;
        tauPrint = NULL;

        delete[] totalDelayPostPrint;
        delete[] tauspost_Print;
        delete[] tauspre_Print;
        delete[] taupos_Print;
        delete[] tauneg_Print;
        delete[] STDPgap_Print;
        delete[] Wex_Print;
        delete[] Aneg_Print;
        delete[] Apos_Print;
        delete[] mupos_Print;
        delete[] muneg_Print;
        delete[] useFroemkeDanSTDP_Print;
        totalDelayPostPrint = NULL;
        tauspost_Print = NULL;
        tauspre_Print = NULL;
        taupos_Print = NULL;
        tauneg_Print = NULL;
        STDPgap_Print = NULL;
        Wex_Print = NULL;
        Aneg_Print = NULL;
        Apos_Print = NULL;
        mupos_Print = NULL;
        muneg_Print = NULL;
        useFroemkeDanSTDP_Print = NULL;

        delete[] lastSpikePrint;
        delete[] rPrint;
        delete[] uPrint;
        delete[] DPrint;
        delete[] UPrint;
        delete[] FPrint;
        lastSpikePrint = NULL;
        rPrint = NULL;
        uPrint = NULL;
        DPrint = NULL;
        UPrint = NULL;
        FPrint = NULL;
    }
}

