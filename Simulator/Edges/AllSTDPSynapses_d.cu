/*
 * AllSTDPSynapses_d.cu
 *
 */

#include "AllSTDPSynapses.h"
#include "AllSpikingSynapses.h"
#include "GPUSpikingModel.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice ) {
	allocSynapseDeviceStruct( allSynapsesDevice, Simulator::getInstance().getTotalNeurons(), Simulator::getInstance().getMaxSynapsesPerNeuron() );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) {
	AllSTDPSynapsesDeviceProperties allSynapsesDeviceProps;

	allocDeviceStruct( allSynapsesDeviceProps, numNeurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllSTDPSynapsesDeviceProperties ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapsesDeviceProps, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::allocDeviceStruct( AllSTDPSynapsesDeviceProperties &allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapses::allocDeviceStruct( allSynapsesDevice, numNeurons, maxSynapsesPerNeuron );

        BGSIZE maxTotalSynapses = maxSynapsesPerNeuron * numNeurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.totalDelayPost_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.delayQueuePost_, maxTotalSynapses * sizeof( BGSIZE ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.delayIndexPost_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.delayQueuePost_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.tauspost_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.tauspre_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.taupos_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.tauneg_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.STDPgap_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.Wex_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.Aneg_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.Apos_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.mupos_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.muneg_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        //HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice.useFroemkeDanSTDP_, maxTotalSynapses * sizeof( bool ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSTDPSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllSTDPSynapsesDeviceProperties allSynapsesDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allSynapsesDeviceProps, allSynapsesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapsesDeviceProps );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSTDPSynapses::deleteDeviceStruct( AllSTDPSynapsesDeviceProperties& allSynapsesDevice ) {
        HANDLE_ERROR( hipFree( allSynapsesDevice.totalDelayPost_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.delayQueuePost_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.delayIndexPost_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.tauspost_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.tauspre_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.taupos_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.tauneg_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.STDPgap_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.Wex_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.Aneg_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.Apos_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.mupos_ ) );
        HANDLE_ERROR( hipFree( allSynapsesDevice.muneg_) );
        //HANDLE_ERROR( hipFree( allSynapsesDevice.useFroemkeDanSTDP_ ) );

        AllSpikingSynapses::deleteDeviceStruct( allSynapsesDevice );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, Simulator::getInstance().getTotalNeurons(), Simulator::getInstance().getMaxSynapsesPerNeuron() );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int numNeurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllSTDPSynapsesDeviceProperties allSynapsesDeviceProps;

        HANDLE_ERROR( hipMemcpy ( &allSynapsesDeviceProps, allSynapsesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allSynapsesDevice, allSynapsesDeviceProps, numNeurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copyHostToDevice( void* allSynapsesDevice, AllSTDPSynapsesDeviceProperties& allSynapsesDeviceProps, int numNeurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSpikingSynapses::copyHostToDevice( allSynapsesDevice, allSynapsesDeviceProps, numNeurons, maxSynapsesPerNeuron );

        BGSIZE maxTotalSynapses = maxSynapsesPerNeuron * numNeurons;
        
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.totalDelayPost_, totalDelayPost_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.delayQueuePost_, delayQueuePost_,
                maxTotalSynapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.delayIndexPost_, delayIndexPost_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.delayQueuePost_, delayQueuePost_,
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.tauspost_, tauspost_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.tauspre_, tauspre_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.taupos_, taupos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.tauneg_, tauneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.STDPgap_, STDPgap_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.Wex_, Wex_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.Aneg_, Aneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.Apos_, Apos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.mupos_, mupos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.muneg_, muneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        //HANDLE_ERROR( hipMemcpy ( allSynapsesDeviceProps.useFroemkeDanSTDP_, useFroemkeDanSTDP_,
                //maxTotalSynapses * sizeof( bool ), hipMemcpyHostToDevice ) ); 
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSTDPSynapses::copySynapseDeviceToHost( void* allSynapsesDevice ) {
	// copy everything necessary
	AllSTDPSynapsesDeviceProperties allSynapsesDeviceProps;

        HANDLE_ERROR( hipMemcpy ( &allSynapsesDeviceProps, allSynapsesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapsesDeviceProps );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  numNeurons            Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copyDeviceToHost( AllSTDPSynapsesDeviceProperties& allSynapsesDevice ) {
        AllSpikingSynapses::copyDeviceToHost( allSynapsesDevice ) ;

	int numNeurons = Simulator::getInstance().getTotalNeurons();
	BGSIZE maxTotalSynapses = Simulator::getInstance().getMaxSynapsesPerNeuron() * numNeurons;

        HANDLE_ERROR( hipMemcpy ( delayQueuePost_, allSynapsesDevice.delayQueuePost_,
                maxTotalSynapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayIndexPost_, allSynapsesDevice.delayIndexPost_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueuePost_, allSynapsesDevice.delayQueuePost_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspost_, allSynapsesDevice.tauspost_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspre_, allSynapsesDevice.tauspre_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( taupos_, allSynapsesDevice.taupos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauneg_, allSynapsesDevice.tauneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( STDPgap_, allSynapsesDevice.STDPgap_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Wex_, allSynapsesDevice.Wex_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Aneg_, allSynapsesDevice.Aneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Apos_, allSynapsesDevice.Apos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( mupos_, allSynapsesDevice.mupos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( muneg_, allSynapsesDevice.muneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
       // HANDLE_ERROR( hipMemcpy ( useFroemkeDanSTDP_, allSynapsesDevice.useFroemkeDanSTDP_,
                //maxTotalSynapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      GPU address of the AllSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  allNeuronsDevice       GPU address of the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  GPU address of the SynapseIndexMap on device memory.
 */
void AllSTDPSynapses::advanceSynapses( void* allSynapsesDevice, void* allNeuronsDevice, void* synapseIndexMapDevice )
{
    int maxSpikes = (int) ((Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate()));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( totalSynapseCount_ + threadsPerBlock - 1 ) / threadsPerBlock;
    // Advance synapses ------------->
    advanceSTDPSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( totalSynapseCount_, (SynapseIndexMap*) synapseIndexMapDevice, g_simulationStep, Simulator::getInstance().getDeltaT(), 
                                (AllSTDPSynapsesDeviceProperties*)allSynapsesDevice, (AllSpikingNeuronsDeviceProperties*)allNeuronsDevice, maxSpikes );
}

/**     
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllSTDPSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllSTDPSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}


/*
 *  Prints GPU SynapsesProps data.
 *   
 *  @param  allSynapsesDeviceProps   GPU address of the corresponding SynapsesDeviceProperties struct on device memory.
 */
void AllSTDPSynapses::printGPUSynapsesProps( void* allSynapsesDeviceProps ) const
{
    AllSTDPSynapsesDeviceProperties allSynapsesProps;

    //allocate print out data members
    BGSIZE size = maxSynapsesPerNeuron_ * countNeurons_;
    if (size != 0) {
        BGSIZE *synapseCountsPrint = new BGSIZE[countNeurons_];
        BGSIZE maxSynapsesPerNeuronPrint;
        BGSIZE totalSynapseCountPrint;
        int countNeuronsPrint;
        int *sourceNeuronIndexPrint = new int[size];
        int *destNeuronIndexPrint = new int[size];
        BGFLOAT *WPrint = new BGFLOAT[size];

        synapseType *typePrint = new synapseType[size];
        BGFLOAT *psrPrint = new BGFLOAT[size];
        bool *inUsePrint = new bool[size];

        for (BGSIZE i = 0; i < size; i++) {
            inUsePrint[i] = false;
        }

        for (int i = 0; i < countNeurons_; i++) {
            synapseCountsPrint[i] = 0;
        }

        BGFLOAT *decayPrint = new BGFLOAT[size];
        int *totalDelayPrint = new int[size];
        BGFLOAT *tauPrint = new BGFLOAT[size];

        int *totalDelayPostPrint = new int[size];
        BGFLOAT *tauspostPrint = new BGFLOAT[size];
        BGFLOAT *tausprePrint = new BGFLOAT[size];
        BGFLOAT *tauposPrint = new BGFLOAT[size];
        BGFLOAT *taunegPrint = new BGFLOAT[size];
        BGFLOAT *STDPgapPrint = new BGFLOAT[size];
        BGFLOAT *WexPrint = new BGFLOAT[size];
        BGFLOAT *AnegPrint = new BGFLOAT[size];
        BGFLOAT *AposPrint = new BGFLOAT[size];
        BGFLOAT *muposPrint = new BGFLOAT[size];
        BGFLOAT *munegPrint = new BGFLOAT[size];
        bool *useFroemkeDanSTDPPrint = new bool[size];

        // copy everything
        HANDLE_ERROR( hipMemcpy ( &allSynapsesProps, allSynapsesDeviceProps, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapseCountsPrint, allSynapsesProps.synapseCounts_, countNeurons_ * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuronPrint = allSynapsesProps.maxSynapsesPerNeuron_;
        totalSynapseCountPrint = allSynapsesProps.totalSynapseCount_;
        countNeuronsPrint = allSynapsesProps.countNeurons_;

        // Set countNeurons_ to 0 to avoid illegal memory deallocation
        // at AllSynapsesProps deconstructor.
        allSynapsesProps.countNeurons_ = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndexPrint, allSynapsesProps.sourceNeuronIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndexPrint, allSynapsesProps.destNeuronIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WPrint, allSynapsesProps.W_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( typePrint, allSynapsesProps.type_, size * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psrPrint, allSynapsesProps.psr_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUsePrint, allSynapsesProps.inUse_, size * sizeof( bool ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( decayPrint, allSynapsesProps.decay_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauPrint, allSynapsesProps.tau_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( totalDelayPrint, allSynapsesProps.totalDelay_, size * sizeof( int ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( totalDelayPostPrint, allSynapsesProps.totalDelayPost_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspostPrint, allSynapsesProps.tauspost_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tausprePrint, allSynapsesProps.tauspre_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauposPrint, allSynapsesProps.taupos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( taunegPrint, allSynapsesProps.tauneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( STDPgapPrint, allSynapsesProps.STDPgap_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WexPrint, allSynapsesProps.Wex_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( AnegPrint, allSynapsesProps.Aneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( AposPrint, allSynapsesProps.Apos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( muposPrint, allSynapsesProps.mupos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( munegPrint, allSynapsesProps.muneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        //HANDLE_ERROR( hipMemcpy ( useFroemkeDanSTDPPrint, allSynapsesProps.useFroemkeDanSTDP_, size * sizeof( bool ), hipMemcpyDeviceToHost ) );

        for(int i = 0; i < maxSynapsesPerNeuron_ * countNeurons_; i++) {
            if (WPrint[i] != 0.0) {
                cout << "GPU W[" << i << "] = " << WPrint[i];
                cout << " GPU sourNeuron: " << sourceNeuronIndexPrint[i];
                cout << " GPU desNeuron: " << destNeuronIndexPrint[i];
                cout << " GPU type: " << typePrint[i];
                cout << " GPU psr: " << psrPrint[i];
                cout << " GPU in_use:" << inUsePrint[i];

                cout << " GPU decay: " << decayPrint[i];
                cout << " GPU tau: " << tauPrint[i];
                cout << " GPU total_delay: " << totalDelayPrint[i];

                cout << " GPU total_delayPost: " << totalDelayPostPrint[i];
                cout << " GPU tauspost_: " << tauspostPrint[i];
                cout << " GPU tauspre_: " << tausprePrint[i];
                cout << " GPU taupos_: " << tauposPrint[i];
                cout << " GPU tauneg_: " << taunegPrint[i];
                cout << " GPU STDPgap_: " << STDPgapPrint[i];
                cout << " GPU Wex_: " << WexPrint[i];
                cout << " GPU Aneg_: " << AnegPrint[i];
                cout << " GPU Apos_: " << AposPrint[i];
                cout << " GPU mupos_: " << muposPrint[i];
                cout << " GPU muneg_: " << munegPrint[i];
               // cout << " GPU useFroemkeDanSTDP_: " << useFroemkeDanSTDPPrint[i] << endl;
            }
        }

        for (int i = 0; i < countNeurons_; i++) {
            cout << "GPU synapse_counts:" << "neuron[" << i  << "]" << synapseCountsPrint[i] << endl;
        }

        cout << "GPU totalSynapseCount:" << totalSynapseCountPrint << endl;
        cout << "GPU maxSynapsesPerNeuron:" << maxSynapsesPerNeuronPrint << endl;
        cout << "GPU countNeurons_:" << countNeuronsPrint << endl;

        // Set countNeurons_ to 0 to avoid illegal memory deallocation
        // at AllDSSynapsesProps deconstructor.
        allSynapsesProps.countNeurons_ = 0;

        delete[] destNeuronIndexPrint;
        delete[] WPrint;
        delete[] sourceNeuronIndexPrint;
        delete[] psrPrint;
        delete[] typePrint;
        delete[] inUsePrint;
        delete[] synapseCountsPrint;
        destNeuronIndexPrint = NULL;
        WPrint = NULL;
        sourceNeuronIndexPrint = NULL;
        psrPrint = NULL;
        typePrint = NULL;
        inUsePrint = NULL;
        synapseCountsPrint = NULL;

        delete[] decayPrint;
        delete[] totalDelayPrint;
        delete[] tauPrint;
        decayPrint = NULL;
        totalDelayPrint = NULL;
        tauPrint = NULL;

        delete[] totalDelayPostPrint;
        delete[] tauspostPrint;
        delete[] tausprePrint;
        delete[] tauposPrint;
        delete[] taunegPrint;
        delete[] STDPgapPrint;
        delete[] WexPrint;
        delete[] AnegPrint;
        delete[] AposPrint;
        delete[] muposPrint;
        delete[] munegPrint;
        delete[] useFroemkeDanSTDPPrint;
        totalDelayPostPrint = NULL;
        tauspostPrint = NULL;
        tausprePrint = NULL;
        tauposPrint = NULL;
        taunegPrint = NULL;
        STDPgapPrint = NULL;
        WexPrint = NULL;
        AnegPrint = NULL;
        AposPrint = NULL;
        muposPrint = NULL;
        munegPrint = NULL;
        useFroemkeDanSTDPPrint = NULL;
    }

}
