#include "ConnGrowth.h"
#include "AllSpikingSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Simulator.h"
#include "Book.h"

/*
 *  Update the weights of the Synapses in the simulation. To be clear,
 *  iterates through all source and destination neurons and updates their
 *  synaptic strengths from the weight matrix.
 *  Note: Platform Dependent.
 *
 *  @param  numNeurons         number of neurons to update.
 *  @param  neurons            The AllNeurons object.
 *  @param  synapses           The AllSynapses object.
 *  @param  allNeuronsDevice   GPU address to the allNeurons struct in device memory.
 *  @param  allSynapsesDevice  GPU address to the allSynapses struct in device memory.
 *  @param  layout             The Layout object.
 */
void ConnGrowth::updateSynapsesWeights(const int numNeurons, IAllNeurons &neurons, IAllSynapses &synapses, AllSpikingNeuronsDeviceProperties* allNeuronsDevice, AllSpikingSynapsesDeviceProperties* allSynapsesDevice, Layout *layout)
{
        Simulator &simulator = Simulator::getInstance();
        // For now, we just set the weights to equal the areas. We will later
        // scale it and set its sign (when we index and get its sign).
        (*W_) = (*area_);

        BGFLOAT deltaT = simulator.getDeltaT();

        // CUDA parameters
        const int threadsPerBlock = 256;
        int blocksPerGrid;

        // allocate device memories
        BGSIZE W_d_size = simulator.getTotalNeurons() * simulator.getTotalNeurons() * sizeof (BGFLOAT);
        BGFLOAT* W_h = new BGFLOAT[W_d_size];
        BGFLOAT* W_d;
        HANDLE_ERROR( hipMalloc ( ( void ** ) &W_d, W_d_size ) );

        neuronType* neuronTypeMapD;
        HANDLE_ERROR( hipMalloc( ( void ** ) &neuronTypeMapD, simulator.getTotalNeurons() * sizeof( neuronType ) ) );

        // copy weight data to the device memory
        for ( int i = 0 ; i < simulator.getTotalNeurons(); i++ )
                for ( int j = 0; j < simulator.getTotalNeurons(); j++ )
                        W_h[i * simulator.getTotalNeurons() + j] = (*W_)(i, j);

        HANDLE_ERROR( hipMemcpy ( W_d, W_h, W_d_size, hipMemcpyHostToDevice ) );

        HANDLE_ERROR( hipMemcpy ( neuronTypeMapD, layout->neuronTypeMap_, simulator.getTotalNeurons() * sizeof( neuronType ), hipMemcpyHostToDevice ) );

        blocksPerGrid = ( simulator.getTotalNeurons() + threadsPerBlock - 1 ) / threadsPerBlock;
        updateSynapsesWeightsDevice <<< blocksPerGrid, threadsPerBlock >>> ( simulator.getTotalNeurons(), deltaT, W_d, simulator.getMaxSynapsesPerNeuron(), allNeuronsDevice, allSynapsesDevice, neuronTypeMapD );

        // free memories
        HANDLE_ERROR( hipFree( W_d ) );
        delete[] W_h;

        HANDLE_ERROR( hipFree( neuronTypeMapD ) );

        // copy device synapse count to host memory
        synapses.copyDeviceSynapseCountsToHost(allSynapsesDevice);
        // copy device synapse summation coordinate to host memory
        synapses.copyDeviceSynapseSumIdxToHost(allSynapsesDevice);
}
